#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int device;
    hipDeviceProp_t properties;

    hipGetDevice(&device); // Get device ID
    hipGetDeviceProperties(&properties, device); // Get device properties

    std::cout << "Maximum threads per block: " << properties.maxThreadsPerBlock << std::endl;

    return 0;
}
