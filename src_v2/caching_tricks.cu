#include "hip/hip_runtime.h"
#include "caching_tricks.hpp"
#include <iostream>
//too confusing dont include #include "matrix_mul.hpp"//order matters this checks for the other header

/*I worked on this kernal a lot specifcly with the memory loading (took around 5 hours...) 
GPT4 came in cluch after a while whe I decided we better make it a SINGLE for loop.
this makes reasoning about stuff much easier. 

all of this work for cach locality tho seems to be in vain since in the end this is slower
*/
static __global__ void sharedMatrixMulKernal(matrix a, matrix b, matrix ans) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    extern __shared__ float sharedA[];

    // Total elements in the submatrix of 'a' to load into shared memory
    int totalElements = blockDim.x * a.cols;

    // Each thread loads one or more elements into shared memory
    for (int idx = threadIdx.x + threadIdx.y * blockDim.x; idx < totalElements; idx += blockDim.x * blockDim.y) {
        int sharedRow = idx / a.cols;
        int sharedCol = idx % a.cols;
        int globalRow = blockIdx.x * blockDim.x + sharedRow;
        if (globalRow < a.rows && sharedCol < a.cols) {
            sharedA[idx] = a.data[globalRow * a.cols + sharedCol];
        }
    }
    __syncthreads();

    // Compute and write back results
    if (row < ans.rows && col < ans.cols) {
        for (int k = 0; k < a.cols; k++) {
            ans.data[row * ans.cols + col] += sharedA[threadIdx.x * a.cols + k] * b.data[k * b.cols + col];
        }
    }
}

matrix sharedMatrixMul(matrix a,matrix b){
    //memory
    int a_size=a.rows*a.cols;
    float * device_a;
    hipMalloc((void**)&device_a, a_size * sizeof(float));
    hipMemcpy(device_a, a.data, a_size* sizeof(float), hipMemcpyHostToDevice);


    int b_size=b.rows*b.cols;
    float * device_b;
    hipMalloc((void**)&device_b, b_size * sizeof(float));
    hipMemcpy(device_b, b.data, b_size* sizeof(float), hipMemcpyHostToDevice);

    int ans_size=a.rows*b.cols;
    float * device_ans;
    hipMalloc((void**)&device_ans, ans_size * sizeof(float));
    hipMemset(device_ans, 0, ans_size * sizeof(float));

    //setup 
    a.data=device_a;
    b.data=device_b;
    matrix ans=(matrix){device_ans,a.rows,b.cols};
    
    constexpr int x=16;
    constexpr int y=16;

    dim3 numThreads(x,y,1);
    dim3 numBlocks((ans.rows+x-1)/x,(ans.cols+x-1)/x,1);
    size_t sharedMemSize = x*a.cols* sizeof(float);

    // std::cout << "threads (" << numThreads.x << ", " << numThreads.y << ", " << numThreads.z << ")"
    //       << " blocks (" << numBlocks.x << ", " << numBlocks.y << ", " << numBlocks.z << ")"
    //       << " mem size " << sharedMemSize << std::endl;

    //ugly fucking hack
    if (sharedMemSize > 48 * 1024) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(sharedMatrixMulKernal), hipFuncAttributeMaxDynamicSharedMemorySize, sharedMemSize);
    }
    sharedMatrixMulKernal<<<numBlocks,numThreads,sharedMemSize>>>(a,b,ans); 
    //collecting
    float * host_ans =(float *)malloc(ans_size * sizeof(float));
    if (!host_ans){
        return {NULL,0,0};
    }

    hipError_t err = hipMemcpy(host_ans, device_ans, ans_size * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        // Handle the error, for example, print the CUDA error string
        printf("\nCUDA Error: %s\n", hipGetErrorString(err));
    }
    ans.data=host_ans;

    //freeing
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_ans);

    return ans;
}