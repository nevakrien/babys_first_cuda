#include "hip/hip_runtime.h"
extern "C" {
#include "matrix_mul.h"
}
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define CAP_SIZE 100

/* 
I set for 3 hours+ on this code because I forgot floats are weird
for the love of god remember to check for margin and not perfection

previous dispair:

this code fails and I  have no idea why
I already mostly eliminated the possibilty its wrong test data. 
since all the other implementations agree on the result
I then started being very paranoid about syncing. then I ran a sanitizer
and also did other checks... nothing.
*/
__global__ void regularMatrixMul(matrix a,matrix b,matrix ans) {
    int row = blockIdx.x;//over simplefied on purpose //* blockDim.y + threadIdx.y;
    int col = blockIdx.y;//over simplefied on purpose //* blockDim.x + threadIdx.x;

    if (row < ans.rows && col < ans.cols) {
        for (int k = 0; k < a.cols; k++) {
            ans.data[row * ans.cols + col] += a.data[row * a.cols + k] * b.data[k * b.cols + col];
        }
    }

}

matrix matrixMulCuda(matrix a,matrix b){
    //memory
    int a_size=a.rows*a.cols;
    float * device_a;
    hipMalloc((void**)&device_a, a_size * sizeof(float));
    hipMemcpy(device_a, a.data, a_size* sizeof(float), hipMemcpyHostToDevice);


    int b_size=b.rows*b.cols;
    float * device_b;
    hipMalloc((void**)&device_b, b_size * sizeof(float));
    hipMemcpy(device_b, b.data, b_size* sizeof(float), hipMemcpyHostToDevice);

    int ans_size=a.rows*b.cols;
    float * device_ans;
    hipMalloc((void**)&device_ans, ans_size * sizeof(float));
    hipMemset(device_ans, 0, ans_size * sizeof(float));

    //setup 
    a.data=device_a;
    b.data=device_b;
    matrix ans=(matrix){device_ans,a.rows,b.cols};

    //run
    hipDeviceSynchronize(); //yet another needless sync in a desprate attempt to fix this code
    dim3 numBlocks(ans.rows,ans.cols,1);
    regularMatrixMul<<<numBlocks,1>>>(a,b,ans); //yes I know I need threads getting to it

    //collecting
    float * host_ans =(float *)malloc(ans_size * sizeof(float));
    if (!host_ans){
        return {NULL,0,0};
    }

    
    hipDeviceSynchronize(); //first paranoia driven needles sync
    hipError_t err = hipMemcpy(host_ans, device_ans, ans_size * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        // Handle the error, for example, print the CUDA error string
        printf("\nCUDA Error: %s\n", hipGetErrorString(err));
    }
    ans.data=host_ans;

    //freeing
    hipFree(device_a);
    hipFree(device_b);
    hipFree(device_ans);

    return ans;
}

int main(){
    int i=0;
    matrix a,b,ans,y;

    FILE *file_a = fopen("matrices_a.bin", "rb");
    FILE *file_b = fopen("matrices_b.bin", "rb");
    FILE *file_ans = fopen("matrices_ans.bin", "rb");

    while(i<CAP_SIZE){
        i+=1;
        printf("\rMultiplying matrix pair %d", i );

        if(readMatrix(file_a,&a)){
            printf("\nread error\n");
            break;
        }

        if(readMatrix(file_b,&b)){
            printf("\nread error\n");
            break;
        }

        if(readMatrix(file_ans,&ans)){
            printf("\nread error\n");
            break;
        }

        y=matrixMulCuda(a,b);
        if(distHeuristic(y,ans)<0.05){
            printf("\nWrong Data\n");
            hipError_t err=hipGetLastError();
            if (err != hipSuccess) {
                // Handle the error, for example, print the CUDA error string
                printf("CUDA Error: %s\n", hipGetErrorString(err));
            }
            return 1;
        }
    }

    printf("\n");

    return 0;
}